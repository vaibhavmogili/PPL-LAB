#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include "hip/hip_runtime.h"


__global__ void add(int *a, int *b, int *c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main()
{
	int N;

	printf("Enter the value of N: ");
	scanf("%d", &N);

	int *A = (int*)malloc(N*sizeof(int));
	int *B = (int*)malloc(N*sizeof(int));
	int *C = (int*)malloc(N*sizeof(int));

	printf("Enter the elements of A: ");
	for(int i=0; i<N; i++)
		scanf("%d", &A[i]);

	printf("Enter the elements of B: ");
	for(int i=0; i<N; i++)
		scanf("%d", &B[i]);

	int *d_A, *d_B, *d_C;
	int size = N * sizeof(int);	

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	add<<<ceil(N/256), 256>>>(d_A, d_B, d_C);

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Array: ");
	for(int i=0; i<N; i++)
		printf("%d ", C[i]);
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}