#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void sin_rad(float *A, float *B)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float val = A[idx];
	float res = sin(val);
	B[idx] = res;
}

int main()
{
	int N;

	printf("Enter the number of elements: ");
	scanf("%d", &N);

	float *A = (float*)malloc(N*sizeof(float));
	float *d_A, *d_B;

	printf("Enter the array elements: ");
	for(int i=0; i<N; i++)
		scanf("%f", &A[i]);

	hipMalloc((void**)&d_A, N*sizeof(float));
	hipMalloc((void**)&d_B, N*sizeof(float));

	hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);

	sin_rad<<<1, N>>>(d_A, d_B);

	hipMemcpy(B, d_B, N*sizeof(float), hipMemcpyDeviceToHost);

	printf("The output array is: ");
	for(int i=0; i<N; i++)
		printf("%.3f ", B[i]);
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);

	return 0;
}