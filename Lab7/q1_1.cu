#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void add(int *a, int *b, int *c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main()
{
	int N;

	printf("Enter the value of N: ");
	scanf("%d", &N);

	int *A = (int*)malloc(N*sizeof(int));
	int *B = (int*)malloc(N*sizeof(int));
	int *C = (int*)malloc(N*sizeof(int));

	printf("Enter the elements of A: ");
	for(int i=0; i<N; i++)
		scanf("%d", &A[i]);

	printf("Enter the elements of B: ");
	for(int i=0; i<N; i++)
		scanf("%d", &B[i]);

	int *d_A, *d_B, *d_C;			// Device buffers
	int size = N * sizeof(int);

	// Allocate memory to device buffers
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	// Copy data to device buffers
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	// First parameter is number of blocks, whereas second parameter is number of threads per block
	add<<<N,1>>>(d_A, d_B, d_C);

	// Copy results back to host
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Array: ");
	for(int i=0; i<N; i++)
		printf("%d ", C[i]);
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}