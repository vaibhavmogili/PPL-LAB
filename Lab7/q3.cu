#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void convolution_1D_mask(int *N, int *M, int *P, int mask_width, int width)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int Pval = 0;
	int N_start_point = idx - (mask_width / 2);

	for(int j=0; j<mask_width; j++)
		if(N_start_point + j >= 0 && N_start_point + j < width)
			Pval += N[N_start_point + j] * M[j];
	P[idx] = Pval;
}

int main()
{
	int width, mask_width;
	int d_width, d_mask_width;

	printf("Enter the width of the input array N: ");
	scanf("%d", &width);
	printf("Enter the width of the mask M: ");
	scanf("%d", &mask_width);

	int *N = (int*)malloc(width*sizeof(int));
	int *P = (int*)malloc(width*sizeof(int));
	int *M = (int*)malloc(mask_width*sizeof(int));
	int *d_N, *d_P, *d_M;

	printf("Enter the elements of N: ");
	for(int i=0; i<width; i++)
		scanf("%d", &N[i]);

	printf("Enter the elements of M: ");
	for(int i=0; i<mask_width; i++)
		scanf("%d", &M[i]);

	hipMalloc((void**)&d_N, width*sizeof(int));
	hipMalloc((void**)&d_P, width*sizeof(int));
	hipMalloc((void**)&d_M, mask_width*sizeof(int));
	hipMalloc((void**)&d_width, sizeof(int));
	hipMalloc((void**)&d_mask_width, sizeof(int));

	hipMemcpy(d_N, N, width*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_P, P, width*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_M, M, mask_width*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_width, width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_mask_width, mask_width, sizeof(int), hipMemcpyHostToDevice);

	convolution_1D_mask<<<1, width>>>(d_N, d_M, d_P, d_mask_width, d_width);

	hipMemcpy(P, d_P, width*sizeof(int), hipMemcpyDeviceToHost);

	printf("The output array P is: ");
	for(int i=0; i<width; i++)
		printf("%d ", P[i]);
	printf("\n");

	hipFree(d_N);
	hipFree(d_P);
	hipFree(d_M);

	return 0;
}