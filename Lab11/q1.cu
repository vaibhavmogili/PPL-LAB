// Matrix Multiplication 2D Grid 2D Block

#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void Matrix_Mult(int *A, int *B, int *C, int ra, int ca, int cb)
{
	int r = blockIdx.x * blockDim.x + threadIdx.x;
	int c = blockIdx.y * blockDim.y + threadIdx.y;

	if(r < ra && c < cb)
	{
		int sum = 0;
		for(int k=0; k<ca; k++)	
			sum += A[r * ca + k] * B[k * cb + c];
		C[r * cb + c] = sum;
	}
}

int main()
{
	int ra, ca, rb, cb;

	printf("Enter the dimensions of matrix A: ");
	scanf("%d %d", &ra, &ca);
	printf("Enter the dimensions of matrix B: ");
	scanf("%d %d", &rb, &cb);

	int *A = (int*)malloc((ra * ca) * sizeof(int));
	int *B = (int*)malloc((rb * cb) * sizeof(int));
	int *C = (int*)malloc((ra * cb) * sizeof(int));

	printf("Enter the elements of matrix A: ");
	for(int i=0; i<(ra * ca); i++)
		scanf("%d", &A[i]);

	printf("Enter the elements of matrix B: ");
	for(int i=0; i<(rb * cb); i++)
		scanf("%d", &B[i]);

	int *d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, (ra * ca) * sizeof(int));
	hipMalloc((void**)&d_B, (rb * cb) * sizeof(int));
	hipMalloc((void**)&d_C, (ra * cb) * sizeof(int));

	hipMemcpy(d_A, A, (ra * ca) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, (rb * cb) * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(ra / 3.0), ceil(cb / 3.0), 1);
	dim3 dimBlock(3, 3, 1);
	Matrix_Mult<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ra, ca, cb);

	hipMemcpy(C, d_C, (ra * cb) * sizeof(int), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Output Matrix:\n");
	for(int i=0; i<(ra * cb); i++)
	{
		if(i % cb == 0)
			printf("\n");
		printf("%d ", C[i]);
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(A);
	free(B);
	free(C);

	return 0;
}