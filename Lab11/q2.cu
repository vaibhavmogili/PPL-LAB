// 1D Convolution Constant Memory

#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


#define K 3

__constant__ int d_kernel[K];

__global__ void Convolution_1D(int *A, int *B)
{
	int tid = threadIdx.x;
	int width = blockDim.x;
	int start_point = tid - (K / 2);
	int sum = 0;

	for(int i=0; i<K; i++)
		if(start_point + i >= 0 && start_point + i < width)
			sum += A[start_point + i] * d_kernel[i];
	B[tid] = sum;
}

int main()
{
	int width;

	printf("Enter width of input array: ");
	scanf("%d", &width);

	int *A = (int*)malloc(width * sizeof(int));
	int *B = (int*)malloc(width * sizeof(int));

	printf("Enter the input array elements: ");
	for(int i=0; i<width; i++)
		scanf("%d", &A[i]);

	int h_kernel[K] = {1, 2, 3};

	int *d_A, *d_B;

	hipMalloc((void**)&d_A, width * sizeof(int));
	hipMalloc((void**)&d_B, width * sizeof(int));
	
	hipMemcpy(d_A, A, width * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, K * sizeof(int));

	Convolution_1D<<<1, width>>>(d_A, d_B);

	hipMemcpy(B, d_B, width * sizeof(int), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Output Array: ");
	for(int i=0; i<width; i++)
		printf("%d ", B[i]);
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	free(A);
	free(B);

	return 0;
}