#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__constant__ int d_kernel[3];
__constant__ int d_A[30];
__constant__ int d_width;
__constant__ int d_mask_width;

__global__ void Constant_Convolution(int *B)
{
	int tid = threadIdx.x;
	int start = tid - (d_mask_width / 2);
	int sum = 0;

	for(int i=0; i<d_mask_width; i++)
		if(start + i >= 0 && start + i < d_width)
			sum += d_A[start + i] * d_kernel[i];
	B[tid] = sum;
}

int main()
{
	int width, mask_width;

	printf("Enter the width of input array: ");
	scanf("%d", &width);
	printf("Enter the width of the mask: ");
	scanf("%d", &mask_width);

	int *A = (int*)malloc(width * sizeof(int));
	int *B = (int*)malloc(width * sizeof(int));
	int *h_kernel = (int*)malloc(mask_width * sizeof(int));

	printf("Enter the input array elements: ");
	for(int i=0; i<width; i++)
		scanf("%d", &A[i]);

	printf("Enter the mask elements: ");
	for(int i=0; i<mask_width; i++)
		scanf("%d", &h_kernel[i]);

	int *d_B;
	hipMalloc((void**)&d_B, width * sizeof(int));

	hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, mask_width * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_A), A, width * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_width), &width, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_mask_width), &mask_width, sizeof(int));

	Constant_Convolution<<<1, width>>>(d_B);

	hipMemcpy(B, d_B, width * sizeof(int), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Output Array : ");
	for(int i=0; i<width; i++)
		printf("%d ", B[i]);
	printf("\n");

	hipFree(d_B);
	free(A);
	free(B);
	free(h_kernel);

	return 0;
}