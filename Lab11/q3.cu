// Tiled Matrix Multiplication 2D Grid 2D Block

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


#define BLOCK_WIDTH 2
#define TILE_WIDTH 2
#define WIDTH 4

__global__ void Tiled_Matrix_Mult(int *A, int *B, int *C)
{
	__shared__ int As[TILE_WIDTH][TILE_WIDTH];
	__shared__ int Bs[TILE_WIDTH][TILE_WIDTH];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = blockIdx.y * TILE_WIDTH + ty;
	int col = blockIdx.x * TILE_WIDTH + tx;
	int res = 0;

	for(int t = 0; t < WIDTH / TILE_WIDTH; t++)
	{
		As[ty][tx] = A[row * WIDTH + t * TILE_WIDTH + tx];
		Bs[ty][tx] = B[(t * TILE_WIDTH + ty) * WIDTH + col];
		__syncthreads();

		for(int k=0; k<TILE_WIDTH; k++)
			res += As[ty][k] * Bs[k][tx];

		__syncthreads();
	}
	C[row * WIDTH + col] = res;
}

int main()
{
	int *A = (int*)malloc((WIDTH * WIDTH) * sizeof(int));
	int *B = (int*)malloc((WIDTH * WIDTH) * sizeof(int));
	int *C = (int*)malloc((WIDTH * WIDTH) * sizeof(int));

	printf("Enter the elements of A: ");
	for(int i=0; i<(WIDTH * WIDTH); i++)
		scanf("%d", &A[i]);

	printf("Enter the elements of B: ");
	for(int i=0; i<(WIDTH * WIDTH); i++)
		scanf("%d", &B[i]);

	int *d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, (WIDTH * WIDTH) * sizeof(int));
	hipMalloc((void**)&d_B, (WIDTH * WIDTH) * sizeof(int));
	hipMalloc((void**)&d_C, (WIDTH * WIDTH) * sizeof(int));

	hipMemcpy(d_A, A, (WIDTH * WIDTH) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, (WIDTH * WIDTH) * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 dimGrid(ceil(WIDTH / BLOCK_WIDTH), ceil(WIDTH / BLOCK_WIDTH), 1);

	Tiled_Matrix_Mult<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	hipMemcpy(C, d_C, (WIDTH * WIDTH) * sizeof(int), hipMemcpyDeviceToHost);

	printf("Output Matrix:\n");
	for(int i=0; i<(WIDTH * WIDTH); i++)
	{
		if(i % WIDTH == 0)
			printf("\n");
		printf("%d ", C[i]);
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(A);
	free(B);
	free(C);

	return 0;
}