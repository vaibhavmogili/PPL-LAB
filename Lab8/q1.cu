#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void Count_Word_Occurrences(char *str, char *word, int str_len, int word_len, int count)
{
	int tid = threadIdx.x;

	while(tid < str_len)
	{
		int i = 0;
		while(i < word_len && str[tid+i] == word[i])
			i++;

		if(i == word_len)
		{
			atomicAdd(count, 1);
			tid += word_len;
		}
		else
			tid++;
	}
}

int main()
{
	char str[100], word[100];
	int str_len, word_len;

	printf("Enter a string: ");
	gets(str);
	str_len = strlen(str);

	printf("Enter a word: ");
	scanf("%s", word);
	word_len = strlen(word);

	char *d_str, *d_word;
	int *d_count, count;

	hipMalloc((void**)&d_str, str_len*sizeof(char));
	hipMalloc((void**)&d_word, word_len*sizeof(char));
	hipMalloc((void**)&d_count, sizeof(int));

	hipMemcpy(d_str, str, str_len*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_word, word, word_len*sizeof(char), hipMemcpyHostToDevice);
	hipMemset(d_count, 0, sizeof(int));

	Count_Word_Occurrences<<<1, 1>>>(d_str, d_word, str_len, word_len, d_count);

	hipMemcpy(count, d_count, sizeof(int), hipMemcpyDeviceToHost);

	printf("Number of Occurrences: %d\n", count);

	hipFree(d_str);
	hipFree(d_word);
	hipFree(d_count);

	return 0;
}