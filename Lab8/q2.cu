#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void Generate_RS(char *S, char *RS, int *idx, int len)
{
	int tid = threadIdx.x;
	for(int i=0; i<len; i++)
		RS[idx[tid]+i] = S[i];
}

int main()
{
	char S[100], RS[100];
	int idx[100];
	int len;

	printf("Enter a word: ")
	scanf("%s", S);
	len = strlen(S);

	idx[0] = 0;
	for(int i=1; i<len; i++)
		idx[i] = idx[i-1] + len - i + 1;

	char *d_S, *d_RS;
	int *d_idx;

	hipMalloc((void**)&d_S, len*sizeof(char));
	hipMalloc((void**))&d_RS, (len*len)*sizeof(char));
	hipMalloc((void**)&d_idx, len*sizeof(int));

	hipMemcpy(d_S, S, len*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_RS, RS, (len*len)*sizeof(char), hipMemcpyHostToDevice);

	Generate_RS<<<1, len>>>(d_S, d_RS, d_idx, len);

	hipMemcpy(RS, d_RS, (len*len)*sizeof(char), hipMemcpyDeviceToHost);

	printf("RS: %s\n", RS);

	hipFree(d_S);
	hipFree(d_RS);
	hipFree(d_idx);

	return 0;
}		