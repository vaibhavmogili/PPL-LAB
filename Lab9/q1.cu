#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void matrix_add_rowwise(int *M1, int *M2, int *res, int c)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i=0; i<c; i++)
		res[tid * c + i] = M1[tid * c + i] + M2[tid * c + i];
}

__global__ void matrix_add_colwise(int *M1, int *M2, int *res, int r)
{
	int tid = threadIdx.x;
	int c = blockDim.x;
	for(int i=0; i<r; i++)
		res[i * c + tid] = M1[i * c + tid] + M2[i * c + tid];
}

__global__ void matrix_add_elementwise(int *M1, int *M2, int *res)
{
	int tid = threadIdx.x;
	res[tid] = M1[tid] + M2[tid];
}

int main()
{
	int r, c;
	int size;

	printf("Enter the number of rows: ");
	scanf("%d", &r);
	printf("Enter the number of columns: ");
	scanf("%d", &c);
	size = r * c * sizeof(int);

	int *M1 = (int*)malloc(size);
	int *M2 = (int*)malloc(size);
	int *res = (int*)malloc(size);

	printf("Enter the elements of M1: ");
	for(int i=0; i<(r*c); i++)
		scanf("%d", &M1[i]);

	printf("Enter the elements of M2: ");
	for(int i=0; i<(r*c); i++)
		scanf("%d", &M2[i]);

	int *d_M1, *d_M2, *d_res;

	hipMalloc((void**)&d_M1, size);
	hipMalloc((void**)&d_M2, size);
	hipMalloc((void**)&d_res, size);

	hipMemcpy(d_M1, M1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_M2, M2, size, hipMemcpyHostToDevice);

	matrix_add_rowwise<<<1, r>>>(d_M1, d_M2, d_res, c);
	// matrix_add_colwise<<<1, c>>>(d_M1, d_M2, d_res, r);
	// matrix_add_elementwise<<<1, r*c>>>(d_M1, d_M2, d_res);

	hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Matrix:\n");
	for(int i=0; i<(r*c); i++)
	{	
		if(i % c == 0)
			printf("\n");
		printf("%d ", res[i]);
	}

	hipFree(d_M1);
	hipFree(d_M2);
	hipFree(d_res);
	free(M1);
	free(M2);
	free(res);

	return 0;
}