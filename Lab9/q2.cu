#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include ""

__global__void matrix_mult_rowwise(int *A, int *B, int *res, int wa, int wb)
{
	int rowidA = threadIdx.x;

	int sum;

	for(int colidB = 0; colidB < wb; colidB++)
	{
		sum = 0;
		for(int k=0; k<wa; k++)
			sum += A[rowidA * wa + k] * B[k * wb + colidB];
		res[rowidA * wb + colidB] = sum;
	}
}

__global__ void matrix_mult_colwise(int *A, int *B, int *res, int wa, int ha)
{
	int colidB = threadIdx.x;
	int sum;
	int wb = blockDim.x;

	for(int rowidA = 0; rowidA < ha; rowidA++)
	{
		sum = 0;
		for(int k=0; k<wa; k++)
			sum += A[rowidA * wa + k] * B[k * wb + colidB];
		res[rowidA * wb + colidB] = sum;
	}
}

__global__ void matrix_mult_elementwise(int *A, int *B, int *res, int wa)
{
	int rowidA = threadIdx.x;
	int colidB = threadIdx.y;
	int sum = 0;
	int wb = blockDim.y;

	for(int k=0; k<wa; k++)
		sum += A[rowidA * wa + k] * B[k * wb + colidB];
	res[rowidA * wb + colidB] = sum;
}

int main()
{
	int r1, c1, r2, c2;

	printf("Enter the dimensions of M1: ");
	scanf("%d %d", &r1, &c1);
	printf("Enter the dimensions of M2: ");
	scanf("%d &d", &r2, &c2);

	if(c1 != r2)
	{
		printf("Invalid Dimensions!\n");
		exit(1);
	}

	int sizeM1 = r1 * c1 * sizeof(int);
	int sizeM2 = r2 * c2 * sizeof(int);
	int sizeRes = r1 * c2 * sizeof(int);

	int *M1 = (int*)malloc(sizeM1);
	int *M2 = (int*)malloc(sizeM2);
	int *res = (int*)malloc(sizeRes);

	printf("Enter the elements of M1: ");
	for(int i=0; i<(r1 * c1); i++)
		scanf("%d", M1[i]);

	printf("Enter the elements of M2: ");
	for(int i=0; i<(r2 * c2); i++)
		scanf("%d", M2[i]);

	int *d_M1, *d_M2, *d_res;

	hipMalloc((void**)&d_M1, sizeM1);
	hipMalloc((void**)&d_M2, sizeM2);
	hipMalloc((void**)&d_res, sizeRes);

	hipMemcpy(d_M1, M1, sizeM1, hipMemcpyHostToDevice);
	hipMemcpy(d_M2, M2, sizeM2, hipMemcpyHostToDevice);

	matrix_mult_rowwise<<<1, r1>>>(d_M1, d_M2, d_res, c1, c2);
	// matrix_mult_colwise<<<1, c2>>>(d_M1, d_M2, d_res, c1, r1);
	// matrix_mult_elementwise<<<1, (r1, c2)>>>(d_M1, d_M2, d_res, c1);

	hipMemcpy(res, d_res, sizeRes, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Matrix:\n");
	for(int i=0; i<(r1*c2); i++)
	{
		if(i % c2 == 0)
			printf("\n");
		printf("%d ", res[i]);
	}

	hipFree(d_M1);
	hipFree(d_M2);
	hipFree(d_res);
	free(M1);
	free(M2);
	free(res);

	return 0;
}