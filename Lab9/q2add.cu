#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void replace_matrix_elements(int *A, int *res, int N)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	int num = A[i * N + j];
	int ans;

	if(i == j)
		ans = 0;
	else if(i < j)
	{
		ans = 0;
		int dig;

		while(num > 0)
		{
			dig = num % 10;
			num /= 10;
			ans += dig;
		}
	}
	else
	{
		ans = 1;
		for(int i=1; i<=num; i++)
			ans *= i;
	}
	res[i * N + j] = ans;
}

int main()
{
	int N;
	int size;

	printf("Enter the value of N: ");
	scanf("%d", &N);
	size = N * N * sizeof(int);

	int *A = (int*)malloc(size);
	int *res = (int*)malloc(size);

	printf("Enter the matrix elements: ");
	for(int i=0; i<(N * N); i++)
		scanf("%d", &A[i]);

	int *d_A, *d_res;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_res, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	replace_matrix_elements<<<1, (N, N)>>>(d_A, d_res, N);

	hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Matrix:\n");
	for(int i=0; i<(N*N); i++)
	{
		if(i % N == 0)
			printf("\n");
		printf("%d ", res[i]);
	}

	hipFree(d_A);
	hipFree(d_res);
	free(A);
	free(res);

	return 0;
}