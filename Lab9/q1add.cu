#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void replace_matrix_elements(int *A, int *res)
{
	int rows = blockDim.x;
	int cols = blockDim.y;
	int row = threadIdx.x;
	int col = threadIdx.y;

	int idx = row * cols + col;
	int sum = 0;

	if(A[idx] % 2 == 0)
		for(int i=0; i<cols; i++)
			sum += A[row * cols + i];
	else
		for(int i=0; i<rows; i++)
			sum += A[i * cols + col];
	res[idx] = sum;
}

int main()
{
	int M, N;
	int size;

	printf("Enter the value of M: ");
	scanf("%d", &M);
	printf("Enter the value of N: ");
	scanf("%d", &N);
	size = M * N * sizeof(int);

	int *A = (int*)malloc(size);
	int *res = (int*)malloc(size);

	printf("Enter the elements of matrix A: ");
	for(int i=0; i<(M*N); i++)
		scanf("%d", &A[i]);

	int *d_A, *d_res;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_res, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	replace_matrix_elements<<<1, (M, N)>>>(d_A, d_res);

	hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Matrix:\n");
	for(int i=0; i<(M*N); i++)
	{
		if(i % M == 0)
			printf("\n");
		printf("%d ", res[i]);
	}

	hipFree(d_A);
	hipFree(d_res);
	free(A);
	free(res);

	return 0;
}