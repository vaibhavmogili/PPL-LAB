#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void Replace_Matrix_Elements(int *A, int *B)
{
	int rowid = threadIdx.x;
	int colid = threadIdx.y;
	int M = blockDim.x;
	int N = blockDim.y;
	int rowsum = 0, colsum = 0, sum;

	for(int i=0; i<N; i++)
		rowsum += A[rowid * N + i];

	for(int i=0; i<M; i++)
		colsum += A[i * N + colid];

	sum = rowsum + colsum;

	B[rowid * N + colid] = sum;
}

int main()
{
	int M, N;
	int size;

	printf("Enter the number of rows: ");
	scanf("%d", &M);
	printf("Enter the number of columns: ");
	scanf("%d", &N);
	size = M * N * sizeof(int);

	int *A = (int*)malloc(size);
	int *B = (int*)malloc(size);

	printf("Enter the elements of A: ");
	for(int i=0; i<(M*N); i++)
		scanf("%d", &A[i]);

	int *d_A, *d_B;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(M, N, 1);
	
	Replace_Matrix_Elements<<<dimGrid, dimBlock>>>(d_A, d_B);

	hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Matrix:\n");
	for(int i=0; i<(M*N); i++)
	{
		if(i % N == 0)
			printf("\n");
		printf("%d ", B[i]);
	}
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	free(A);
	free(B);

	return 0;
}