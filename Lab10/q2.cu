#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void Replace_Matrix_Elements(int *A, int N)
{
	int row = threadIdx.x;

	for(int i=0; i<N; i++)
	{
		int elem = A[row * N + i];
		int temp = elem;
		for(int j=0; j<row; j++)
			elem *= temp;
		A[row * N + i] = elem;
	}
}

int main()
{
	int M, N;
	int size;

	printf("Enter the number of rows: ");
	scanf("%d", &M);
	printf("Enter the number of columns: ");
	scanf("%d", &N);
	size = M * N * sizeof(int);

	int *A = (int*)malloc(size);

	printf("Enter the elements of A: ");
	for(int i=0; i<(M*N); i++)
		scanf("%d", &A[i]);

	int *d_A;

	hipMalloc((void**)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	Replace_Matrix_Elements<<<1, M>>>(d_A, N);

	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Matrix:\n");
	for(int i=0; i<(M*N); i++)
	{
		if(i % N == 0)
			printf("\n");
		printf("%d ", A[i]);
	}
	printf("\n");

	hipFree(d_A);
	free(A);

	return 0;
}