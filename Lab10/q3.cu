#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void Replace_Matrix_Elements(int *A, int *B)
{
	int rowid = threadIdx.x;
	int colid = threadIdx.y;
	int rows = blockDim.x;
	int cols = blockDim.y;
	int binary = 0, base = 1;
	int res = 0, cnt = 1;
	int temp, elem = A[rowid * cols + colid];;

	if(rowid == 0 || colid == 0 || rowid == (cols-1) || colid == (rows-1))
		res = elem;
	else
	{
		while(elem > 0)
		{
			binary += (elem % 2) * base;
			elem /= 2;
			base *= 10;
		}

		while(binary > 0)
		{
			temp = binary % 10;
			if(temp == 0)
				res = (1 * cnt) + res;
			else
				res = (0 * cnt) + res;
			binary /= 10;
			cnt *= 10;
		}
	}
	B[rowid * cols + colid] = res;
}

int main()
{
	int M, N;
	int size;

	printf("Enter the number of rows: ");
	scanf("%d", &M);
	printf("Enter the number of columns: ");
	scanf("%d", &N);
	size = M * N * sizeof(int);

	int *A = (int*)malloc(size);
	int *B = (int*)malloc(size);

	printf("Enter the elements of A: ");
	for(int i=0; i<(M*N); i++)
		scanf("%d", &A[i]);

	int *d_A, *d_B;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(M, N, 1);
	Replace_Matrix_Elements<<<dimGrid, dimBlock>>>(d_A, d_B);

	hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Matrix:\n");
	for(int i=0; i<(M*N); i++)
	{
		if(i % N == 0)
			printf("\n");
		printf("%d ", B[i]);
	}
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	free(A);
	free(B);

	return 0;
}