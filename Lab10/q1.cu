#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void SpVM_CSR(int no_rows, int *data, int *col_idx, int *row_ptr, int *X, int *Y)
{
	int row = threadIdx.x;

	if(row < no_rows)
	{
		int dot = 0;
		int row_start = row_ptr[row];
		int row_end = row_ptr[row+1];
		for(int i=row_start; i<row_end; i++)
			dot += data[i] * X[col_idx[i]];
		Y[row] += dot;
	}
}

int main()
{
	int r, c;

	printf("Enter the number of rows: ");
	scanf("%d", &r);
	printf("Enter the number of columns: ");
	scanf("%d", &c);

	int **A = (int**)malloc(r * sizeof(int*));
	for(int i=0; i<r; i++)
		A[i] = (int*)malloc(c * sizeof(int));

	printf("Enter the matrix elements:\n");
	for(int i=0; i<r; i++)
		for(int j=0; j<c; j++)
			scanf("%d", &A[i][j]);

	int *X = (int*)malloc(c * sizeof(int));
	int *Y = (int*)malloc(r * sizeof(int));

	printf("Enter the elements of vector X: ");
	for(int i=0; i<c; i++)
		scanf("%d", &X[i]);

	int data[20], col_idx[20], row_ptr[20];
	int count = 0, temp;
	row_ptr[0] = 0;
	int i;

	for(i=0; i<r; i++)
	{
		temp = 0;
		for(int j=0; j<c; j++)
		{
			if(A[i][j] != 0)
			{
				temp++;
				data[count] = A[i][j];
				col_idx[count] = j;
				count++;
			}
		}
		row_ptr[i+1] = row_ptr[i] + temp;
	}

	int *d_data, *d_row_ptr, *d_col_idx, *d_X, *d_Y;

	hipMalloc((void**)&d_data, count * sizeof(int));
	hipMalloc((void**)&d_col_idx, count * sizeof(int));
	hipMalloc((void**)&d_row_ptr, (r + 1) * sizeof(int));
	hipMalloc((void**)&d_X, c * sizeof(int));
	hipMalloc((void**)&d_Y, r * sizeof(int));

	hipMemcpy(d_data, data, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_col_idx, col_idx, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row_ptr, row_ptr, (r + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_X, X, c * sizeof(int), hipMemcpyHostToDevice);

	SpVM_CSR<<<1, r>>>(r, d_data, d_col_idx, d_row_ptr, d_X, d_Y);

	hipMemcpy(Y, d_Y, r * sizeof(int), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Y : ");
	for(int j=0; j<r; j++)
		printf("%d ", Y[j]);
	printf("\n");

	hipFree(d_data);
	hipFree(d_col_idx);
	hipFree(d_row_ptr);
	hipFree(d_X);
	hipFree(d_Y);
	free(A);

	return 0;
}