#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void Generate_STR(char *A, int *B, int *idx, char *res)
{
	int rowid = threadIdx.x;
	int colid = threadIdx.y;
	int N = blockDim.y;

	char ch = A[rowid * N + colid];
	int n = B[rowid * N + colid];
	int start = idx[rowid * N + colid];

	for(int i=0; i<n; i++)
		res[start+i] = ch;
}

int main()
{
	int M, N;

	printf("Enter the number of rows: ");
	scanf("%d", &M);
	printf("Enter the number of columns: ");
	scanf("%d", &N);

	char *A = (char*)malloc(M * N * sizeof(char));
	int *B = (int*)malloc(M * N * sizeof(int));

	printf("Enter the characters of A: ");
	for(int i=0; i<(M*N); i++)
		scanf(" %c", &A[i]);

	printf("Enter the numbers of B: ");
	for(int i=0; i<(M*N); i++)
		scanf("%d", &B[i]);

	int size = 0;
	for(int i=0; i<(M*N); i++)
		size += B[i];
	char *res = (char*)malloc(size * sizeof(char));

	int *idx = (int*)malloc(M * N * sizeof(int));
	idx[0] = 0;

	for(int i=1; i<(M*N); i++)
		idx[i] = idx[i-1] + B[i-1];

	char *d_A;
	int *d_B, *d_idx;
	char *d_res;

	hipMalloc((void**)&d_A, M * N * sizeof(char));
	hipMalloc((void**)&d_B, M * N * sizeof(int));	
	hipMalloc((void**)&d_idx, M * N * sizeof(int));
	hipMalloc((void**)&d_res, size * sizeof(char));

	hipMemcpy(d_A, A, M * N * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, M * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_idx, idx, M * N * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(M, N, 1);

	Generate_STR<<<dimGrid, dimBlock>>>(d_A, d_B, d_idx, d_res);

	hipMemcpy(res, d_res, size * sizeof(char), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("STR : ");
	for(int i=0; i<size; i++)
		printf("%c", res[i]);
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_idx);
	hipFree(d_res);
	free(A);
	free(B);
	free(idx);
	free(res);

	return 0;
}